#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "cuda/hip/hip_runtime_api.h"
#include <cuda/Orb.hpp>

using namespace cv;
using namespace cv::cuda;
using namespace cv::cuda::device;

namespace ORB_SLAM3::cuda::orb {

  __constant__ unsigned char c_pattern[sizeof(Point) * 512];

  void GpuOrb::loadPattern(const Point * _pattern) {
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c_pattern), _pattern, sizeof(Point) * 512) );
  }

  __device__ int getOrbValue(const PtrStepb image, const short2 loc, const Point* pattern, int idx, float a, float b) {

    return static_cast<int>(image(loc.y + __float2int_rn(pattern[idx].x * b + pattern[idx].y * a),
          loc.x + __float2int_rn(pattern[idx].x * a - pattern[idx].y * b)));
  }

  __global__ void calcOrb_kernel(const PtrStepb image, KeyPoint * keypoints, const int npoints, PtrStepb descriptors) {
    int id = blockIdx.x;
    int tid = threadIdx.x;
    if (id >= npoints) return;

    const KeyPoint &kpt = keypoints[id];
    short2 loc = make_short2(kpt.pt.x, kpt.pt.y);
    const Point * pattern = ((Point *)c_pattern) + 16 * tid;

    uchar * desc = descriptors.ptr(id);
    const float factorPI = (float)(CV_PI/180.f);
    float angle = (float)kpt.angle * factorPI;
    float a = (float)cosf(angle), b = (float)sinf(angle);

    int t0, t1, val;
    t0 = getOrbValue(image, loc, pattern, 0, a, b); t1 = getOrbValue(image, loc, pattern, 1, a, b);
    val = t0 < t1;
    t0 = getOrbValue(image, loc, pattern, 2, a, b); t1 = getOrbValue(image, loc, pattern, 3, a, b);
    val |= (t0 < t1) << 1;
    t0 = getOrbValue(image, loc, pattern, 4, a, b); t1 = getOrbValue(image, loc, pattern, 5, a, b);
    val |= (t0 < t1) << 2;
    t0 = getOrbValue(image, loc, pattern, 6, a, b); t1 = getOrbValue(image, loc, pattern, 7, a, b);
    val |= (t0 < t1) << 3;
    t0 = getOrbValue(image, loc, pattern, 8, a, b); t1 = getOrbValue(image, loc, pattern, 9, a, b);
    val |= (t0 < t1) << 4;
    t0 = getOrbValue(image, loc, pattern, 10, a, b); t1 = getOrbValue(image, loc, pattern, 11, a, b);
    val |= (t0 < t1) << 5;
    t0 = getOrbValue(image, loc, pattern, 12, a, b); t1 = getOrbValue(image, loc, pattern, 13, a, b);
    val |= (t0 < t1) << 6;
    t0 = getOrbValue(image, loc, pattern, 14, a, b); t1 = getOrbValue(image, loc, pattern, 15, a, b);
    val |= (t0 < t1) << 7;

    desc[tid] = (uchar)val;



  }



  GpuOrb::GpuOrb(int maxKeypoints) : maxKeypoints(maxKeypoints), descriptors(maxKeypoints, 32, CV_8UC1) {
    checkCudaErrors( hipStreamCreate(&stream) );
    cvStream = StreamAccessor::wrapStream(stream);
    checkCudaErrors( hipMalloc(&keypoints, sizeof(KeyPoint) * maxKeypoints) );
  }

  GpuOrb::~GpuOrb() {
    cvStream.~Stream();
    checkCudaErrors( hipFree(keypoints) );
    checkCudaErrors( hipStreamDestroy(stream) );
  }

  hipStream_t GpuOrb::getStream(){
    return stream;
  }

  void GpuOrb::launch_async(InputArray _image, const KeyPoint * _keypoints, const int npoints) {
    if (npoints == 0) {
      return ;
    }
    const GpuMat image = _image.getGpuMat();

    checkCudaErrors( hipMemcpyAsync(keypoints, _keypoints, sizeof(KeyPoint) * npoints, hipMemcpyHostToDevice, stream) );
    desc = descriptors.rowRange(0, npoints);
    desc.setTo(Scalar::all(0), cvStream);

    dim3 dimBlock(32);
    dim3 dimGrid(npoints);
    calcOrb_kernel<<<dimGrid, dimBlock, 0, stream>>>(image, keypoints, npoints, desc);
    checkCudaErrors( hipGetLastError() );
  }

  void GpuOrb::join(Mat & _descriptors) {
    desc.download(_descriptors, cvStream);
    checkCudaErrors( hipStreamSynchronize(stream) );
  }
}
